#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

#define MAX_SHARED_ELEMS 12*1024
#define MAX_LENGTH 512*1024*1024
#define MAX_LENGTH_SEQ 1*1024*1024
#define REPETITIONS 5


// UTILS
void randomArray(int *a, int n){
/*
    Randomly initialize the elements of an array given its size.
*/
    for(int i=0; i < n; i++){
        a[i] = rand();
    }
}


void copyArray(int *a, int *b, int n){
/*
    Copy elements from one array to another.
*/
    for(int i=0; i < n; i++){
        b[i] = a[i];
    }
}


__global__ void copyArrayDevice(int *dst, int *src, int n, int blocks){
/*
    Copy elements from one array in the DEVICE memory into another one in the DEVICE memory.
*/
    int threads = blocks * blockDim.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i=idx; i < n; i+=threads){
        dst[i] = src[i];
    }
}


bool compareArrays(int *a, int *b, int n){
/*
    Compare the elements of two arrays.
*/
    for(int i=0; i < n; i++){
        if(a[i] != b[i]){
            return false;
        }
    }
    return true;
}


bool everIncreasing(int *a, int n){
/*
    Check wheter the array is monotonically increasing.
*/
    for(int i=1; i < n; i++){
        if(a[i] < a[i-1]){
            return false;
        }
    }
    return true;
}


void printResults(double *time, double *thr, int n){
    for(int r=0; r < n; r++){
        printf("%.4f,    ", time[r]);
    }
    printf("(s)\n");
    for(int r=0; r < n; r++){
        printf("%.0f,    ", thr[r]);
    }
    printf("(el/s)\n\n");
}


__host__ __device__ int binarySearchCount(int val, int *a, int first, int last, int ord){
/*
    Count the number of elements that are smaller than 'val' (also equal if 'ord' is 1).
*/
    int left = first;
    int right = last;
    int mid;
    while(left < right){
        mid = (right + left) / 2;
        
        if(a[mid] == val && ord == 1){
            while(mid + 1 < last && a[mid+1] == val){
                mid++;
            }
            return mid + 1 - first;
        }
            
        if(a[mid] == val && ord == 0){
            while(mid - 1 >= first && a[mid-1] == val){
                mid--;
            }
            return mid - first;       
        }

        if(a[mid] > val){
            right = mid;
        } else {
            left = mid + 1;
        }
    }
    while(mid >= first && a[mid] > val){
        mid--;
    }
    return mid + 1 - first;
}


// SEQUENTIAL VERSION
void mergeSortSeq(int *a, int n){
    int *aux = (int *)malloc(n * sizeof(int));
    int ord, before, first, last;

    // Sub-arrays
    for(int size=1; size < n; size*=2){

        // Element sorting
        for(int i=0; i < n; i++){
            ord = (i/size) % 2;                             // Indicates if first or second array of the pair
            before = i%size + (i/(2*size)) * 2 * size;      // N° of elements before the current pair of sub-arrays
            first = (i/size + 1 - 2*ord) * size;            // Index of the first element of the sub-array used for binary search
            last = min(first + size, n);                    // Index of the last element of the sub-array used for binary search

            // Copy the elements in the right position
            if(last > first){
                int idx = binarySearchCount(a[i], a, first, last, ord);
                aux[idx + before] = a[i];
            } else {
                aux[before] = a[i];
            }
        }
        // Move the values back from the auxiliary array into the original array
        copyArray(aux, a, n);
    }
}


// PARALLEL VERSION
__global__ void mergeInMem(int *a, int n, int blocks){
/*
    Performs multiple steps of merge sort, until data does not fit the shared memory anymore.

    In these kernel all the computations are performed using the shared memory.
*/ 

    int elems_block = (n + blocks - 1) / blocks;    // Elements per block
    int off = blockIdx.x * elems_block;             // Block offset

    // Shared memory
    __shared__ int v[MAX_SHARED_ELEMS/2];
    __shared__ int aux[MAX_SHARED_ELEMS/2];

    // Fill shared memory: each block loads the values in a local vector 'v'
    for(int i=threadIdx.x; (i < elems_block) && (i + off < n); i+=blockDim.x){
        v[i] = a[i+off];
    }

    int ord, before, first, last;
    for(int size=1; size < elems_block; size*=2){
        __syncthreads();
        for(int i=threadIdx.x; i < elems_block && i + off < n; i+=blockDim.x){
            ord = (i/size) % 2;                             // Indicates if first or second array of the pair
            before = i%size + (i/(2*size)) * 2 * size;      // N° of elements before the current pair of sub-arrays 
            first = (i/size + 1 - 2*ord) * size;            // Index of the first element of the sub-array used for binary search
            last = min(first + size, elems_block);          
            last = min(last, n - off);                      // Index of the last element of the sub-array used for binary search
            
            // Copy the elements in the right position
            if(last > first){
                int idx = binarySearchCount(v[i], v, first, last, ord);
                aux[idx + before] = v[i]; 
            } else {
                aux[before] = v[i];
            }
            
        }
        __syncthreads();
        // Move the values back from the auxiliary array into the original array
        for(int i=threadIdx.x; i < elems_block && i + off < n; i+=blockDim.x){
            v[i] = aux[i]; 
        }
    }

    // Write back the data from shared to global memory
    for(int i=threadIdx.x; (i < elems_block) && (i + off < n); i+=blockDim.x){
        a[i+off] = v[i];
    }
}


__global__ void mergeOffMem(int *a, int *aux, int n, int size, int blocks){
/*
    Performs one step of merge sort. 
    In particular, it computes the step in which the sub-arrays have a numer of elements equal to 'size'.

    In these kernel all the computations are performed using the global memory.
*/ 
    int elems_block = (n + blocks - 1) / blocks;    // Elements per block
    int off = blockIdx.x * elems_block;             // Block offset

    int i, ord, before, first, last; 
    for(int j=threadIdx.x; (j < elems_block) && (j + off < n); j+=blockDim.x){
        i = j + off;
        ord = (i/size) % 2;                             // Indicates if first or second array of the pair
        before = i%size + (i/(2*size)) * 2 * size;      // N° of elements before the current pair of sub-arrays
        first = (i/size + 1 - 2*ord) * size;            // Index of the first element of the sub-array used for binary search
        last = min(first + size, n);                    // Index of the last element of the sub-array used for binary search
        
        // Copy the elements in the right position
        if(last > first){
            int idx = binarySearchCount(a[i], a, first, last, ord);
            aux[idx + before] = a[i];
        } else {
            aux[before] = a[i];
        }
    }
}


void mergeSortPar(int *a, int n){
/*
    Merge sort is a sorting algorithm with log_2(n) merging steps.
    
    Advantages:
        - The sequential version is already very efficient.
        - Parallel time complexity: θ( (n/p) * log(n) )

    Disadvantages:
        - The original merge function is not good for parallel implementations (here binary search is used).
        - Not in-place (requires an auxiliary array -> doubles the memory requirements)
*/

    int *d_a, *d_aux;
	const size_t size = n * sizeof(int);
    
    hipMalloc((void **)&d_a, size);      // Allocate space in DEVICE memory
    hipMalloc((void **)&d_aux, size);    // Allocate space in DEVICE memory

    // Copy the auxiliary array into the DEVICE 
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    
    // Use the number of blocks that maximize the utilization of shared memory
    int blocks = (n + double(MAX_SHARED_ELEMS/2) - 1) / double(MAX_SHARED_ELEMS/2);
    int threads = 1024;
    int elems_block = (n + blocks - 1) / blocks;
    
    // Initial merging steps using the SHARED memory
    mergeInMem<<<blocks, threads>>>(d_a, n, blocks);
    hipDeviceSynchronize(); // Wait for all the blocks
    for(int size=elems_block; size < n; size*=2){
        
        // Continue merging using the GLOBAL memory
        mergeOffMem<<<blocks, threads>>>(d_a, d_aux, n, size, blocks);
        hipDeviceSynchronize(); // Wait for all the blocks

        // Copy data from the auxiliary array to the original one
        copyArrayDevice<<<blocks, threads>>>(d_a, d_aux, n, blocks);
        hipDeviceSynchronize(); // Wait for all the blocks
    }
   
    // Copy the results from the array in the DEVICE to the one in the HOST memory
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);

    // Free DEVICE memory
    hipFree(d_a); hipFree(d_aux);
}


// MAIN
int main(){

    int *a, *b;
    clock_t start, stop;
    double timeS[REPETITIONS], timeP[REPETITIONS];
    double thrS[REPETITIONS], thrP[REPETITIONS];

    for(int length=1024; length <= MAX_LENGTH; length*=2){
        printf("\n\n############################################");
        printf("############################################\n\n");
        printf("N = %d\n\n", length);
        for(int r=0; r < REPETITIONS; r++){
            
            // Allocate memory for the array
            a = (int *)malloc(length * sizeof(int));
            b = (int *)malloc(length * sizeof(int));

            // Initialize random array
            randomArray(a, length);
            copyArray(a, b, length);

            if(length <= MAX_LENGTH_SEQ){
                // SEQUENTIAL algorithm
                start = clock();
                mergeSortSeq(a, length);
                stop = clock();
            }

            timeS[r] = double(stop-start+1) / double(CLOCKS_PER_SEC);
            thrS[r] = double(length * (double(CLOCKS_PER_SEC) / double(stop-start+1)));

            // PARALLEL algorithm
            start = clock();
            mergeSortPar(b, length);
            stop = clock();

            timeP[r] = double(stop-start+1) / double(CLOCKS_PER_SEC);
            thrP[r] = double(length * (double(CLOCKS_PER_SEC) / double(stop-start+1)));

            // Check correctness
            if((length <= MAX_LENGTH_SEQ && compareArrays(a, b, length) != true) ||
                    (length > MAX_LENGTH_SEQ && everIncreasing(b, length) != true)){
                printf("\nERROR!!\n");
            }

            free(a); free(b);
        }
        if(length <= MAX_LENGTH_SEQ){
            printf("SEQ\n");
            printResults(timeS, thrS, REPETITIONS);
        }
        printf("PAR\n");
        printResults(timeP, thrP, REPETITIONS);
    }
}
